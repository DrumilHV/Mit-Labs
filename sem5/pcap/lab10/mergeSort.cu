#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 16
#define THREADS_PER_BLOCK 256

__device__ void merge(int* d_data, int* d_result, int left, int mid, int right) {
    __shared__ int s_data[N];
    int i = left;
    int j = mid + 1;
    int k = threadIdx.x;

    while (i <= mid && j <= right) {
        if (d_data[i] < d_data[j]) {
            s_data[k] = d_data[i];
            i++;
        } else {
            s_data[k] = d_data[j];
            j++;
        }
        k += blockDim.x;
    }

    while (i <= mid) {
        s_data[k] = d_data[i];
        i++;
        k += blockDim.x;
    }

    while (j <= right) {
        s_data[k] = d_data[j];
        j++;
        k += blockDim.x;
    }

    k = threadIdx.x;
    for (int idx = left; idx <= right; idx++) {
        d_result[idx] = s_data[k];
        k += blockDim.x;
    }
}

__global__ void mergeSort(int* d_data, int* d_result, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int left, mid, right;

    for (int width = 2; width < size; width <<= 1) {
        left = tid * width;
        mid = left + width / 2 - 1;
        right = min(left + width - 1, size - 1);

        if (left < size) {
            merge(d_data, d_result, left, mid, right);
        }
        __syncthreads();

        // Swap data pointers to update the result for the next iteration
        int* temp = d_data;
        d_data = d_result;
        d_result = temp;
    }
}

int main() {
    int h_data[N];
    int h_result[N];
    int* d_data;
    int* d_result;

    // Generate or load your data into h_data array
    for (int i = 0; i < N; i++) {
        h_data[i] = N-i;
        printf("%2d ", h_data[i]);
    }

    hipMalloc((void**)&d_data, N * sizeof(int));
    hipMalloc((void**)&d_result, N * sizeof(int));

    hipMemcpy(d_data, h_data, N * sizeof(int), hipMemcpyHostToDevice);

    mergeSort<<<1, THREADS_PER_BLOCK>>>(d_data, d_result, N);

    hipDeviceSynchronize(); // Wait for the sorting to complete

    hipMemcpy(h_result, d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the sorted data
    printf("Sorted Data:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_result[i]);
    }
    printf("\n");

    hipFree(d_data);
    hipFree(d_result);

    return 0;
}
