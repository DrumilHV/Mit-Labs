#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 16 // Input signal size
#define M 5    // Filter size

__constant__ float d_Filter[M]; // Declare filter coefficients in constant memory

__global__ void convolution(const float *d_Input, float *d_Output) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float result = 0.0f;

    for (int i = 0; i < M; i++) {
        int idx = tid - (M / 2) + i;
        if (idx >= 0 && idx < N) {
            result += d_Input[idx] * d_Filter[i];
        }
    }

    d_Output[tid] = result;
}

int main() {
    float *h_Input, *h_Output;
    float *d_Input, *d_Output;

    // Initialize host arrays
    h_Input = (float*)malloc(N * sizeof(float));
    h_Output = (float*)malloc(N * sizeof(float));

    // Initialize input signal with dummy data
    for (int i = 0; i < N; i++) {
        h_Input[i] = (float)(i+1); // Assign 1.0 to each element as dummy data
    }

    // Initialize filter coefficients in host constant memory
    float h_Filter[M] = {1.0, 2.0, 3.0, 2.0, 1.0};
    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, M * sizeof(float)); // Copy filter to constant memory

    // Allocate device memory
    hipMalloc((void**)&d_Input, N * sizeof(float));
    hipMalloc((void**)&d_Output, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_Input, h_Input, N * sizeof(float), hipMemcpyHostToDevice);

    // Define thread block and grid dimensions
    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Launch the convolution kernel
    convolution<<<numBlocks, threadsPerBlock>>>(d_Input, d_Output);

    // Copy results back from device to host
    hipMemcpy(h_Output, d_Output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < N; i++) {
        printf(" %.2f ", h_Output[i]);
    }

    // Free device and host memory
    hipFree(d_Input);
    hipFree(d_Output);
    free(h_Input);
    free(h_Output);

    return 0;
}
