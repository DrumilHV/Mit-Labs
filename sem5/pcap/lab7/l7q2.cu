
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define matrix dimensions
#define N 4
#define M 4
#define K 4

// CUDA kernel for matrix multiplication where each row is computed by one thread
__global__ void matrixMultRow(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += a[row * K + k] * b[k * M + col];
        }
        c[row * M + col] = sum;
    }
}

// CUDA kernel for matrix multiplication where each column is computed by one thread
__global__ void matrixMultColumn(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += a[row * K + k] * b[k * M + col];
        }
        c[row * M + col] = sum;
    }
}

// CUDA kernel for matrix multiplication where each element is computed by one thread
__global__ void matrixMultElement(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += a[row * K + k] * b[k * M + col];
        }
        c[row * M + col] = sum;
    }
}

int main() {
    float *h_a, *h_b, *h_c; // Host matrices
    float *d_a, *d_b, *d_c; // Device matrices

    // Allocate memory on the host
    h_a = (float *)malloc(N * K * sizeof(float));
    h_b = (float *)malloc(K * M * sizeof(float));
    h_c = (float *)malloc(N * M * sizeof(float));

    // Initialize host matrices
    for (int i = 0; i < N * K; i++) {
        h_a[i] = 1.0f;
    }
    for (int i = 0; i < K * M; i++) {
        h_b[i] = 2.0f;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_a, N * K * sizeof(float));
    hipMalloc((void **)&d_b, K * M * sizeof(float));
    hipMalloc((void **)&d_c, N * M * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, K * M * sizeof(float), hipMemcpyHostToDevice);

    // Define thread and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + 15) / 16, (N + 15) / 16);

    // Launch kernels
    matrixMultRow<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
    //matrixMultColumn<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);
    //matrixMultElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(h_c, d_c, N * M * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result of Matrix Multiplication:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("%.2f\t", h_c[i * M + j]);
        }
        printf("\n");
    }

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
