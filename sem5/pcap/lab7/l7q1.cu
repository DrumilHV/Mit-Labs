
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Define matrix dimensions
#define N 4
#define M 4

// CUDA kernel for adding two matrices where each row is computed by one thread
__global__ void addMatricesRow(float *a, float *b, float *c) {
    int row = threadIdx.x;
    for (int col = 0; col < M; col++) {
        int idx = row * M + col;
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for adding two matrices where each column is computed by one thread
__global__ void addMatricesColumn(float *a, float *b, float *c) {
    int col = threadIdx.x;
    for (int row = 0; row < N; row++) {
        int idx = row * M + col;
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for adding two matrices where each element is computed by one thread
__global__ void addMatricesElement(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < M) {
        int idx = row * M + col;
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c; // Host matrices
    float *d_a, *d_b, *d_c; // Device matrices

    // Allocate memory on the host
    h_a = (float *)malloc(N * M * sizeof(float));
    h_b = (float *)malloc(N * M * sizeof(float));
    h_c = (float *)malloc(N * M * sizeof(float));

    // Initialize host matrices
    for (int i = 0; i < N * M; i++) {
        // h_a[i] = 1.0f;
        // h_b[i] = 2.0f;
        h_a[i] = rand()%100;
        h_b[i] = rand()%100;
    }
    printf("This is matrix A: \n");
    for(int i=0;i<N;i++){
        for(int j=0;j<M;j++){
            printf("%.3f ", h_a[i*M +j]);
        }
        printf("\n");
    }
    printf("This is matrix B: \n");
    for(int i=0;i<N;i++){
        for(int j=0;j<M;j++){
            printf("%.3f ", h_b[i*M +j]);
        }
        printf("\n");
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_a, N * M * sizeof(float));
    hipMalloc((void **)&d_b, N * M * sizeof(float));
    hipMalloc((void **)&d_c, N * M * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * M * sizeof(float), hipMemcpyHostToDevice);

    // Define thread and block dimensions for element-wise addition
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (M + 15) / 16);

    // Launch kernels
    addMatricesRow<<<1, N>>>(d_a, d_b, d_c);
    addMatricesColumn<<<1, M>>>(d_a, d_b, d_c);
    addMatricesElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(h_c, d_c, N * M * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result (for the element-wise addition)
    printf("Result of Element-Wise Addition:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("%.3f\t", h_c[i * M + j]);
        }
        printf("\n");
    }

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
