#include <stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <time.h>

#define M 4
#define N 4
__global__ void addMatrixRow(int *d_a, int *d_b,int *d_c){
	int row = threadIdx.x;
	if(row<M){
		for(int col=0;col<N;col++){
			int idx = row * M + col;
			d_c[idx] = d_b[idx] + d_a[idx];
		}
	}
		
}
__global__ void addMatrixCol(int *d_a, int *d_b,int *d_c){
	int col = threadIdx.x;
	if(col<N){
		for(int row=0;row<M;row++){
			int idx = row * N + col;
			d_c[idx] = d_b[idx] + d_a[idx];
		}
	}		
}
__global__ void addMatrixEle(int *d_a, int *d_b,int *d_c){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row<M && col<N){
		int idx = row * M + col;
		d_c[idx] = d_b[idx]+d_a[idx];
	}	
}
void printFn(int *mat,char c){
	printf("This is matrix %c: \n",c);
	// printf("\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf("%3d ",mat[i*M+j]);
		}
		printf("\n");
	}
	printf("\n");
}

int main(){
	int *h_a,*h_b, *h_c;
	int *d_a, *d_b, *d_c;

	h_a = (int *)malloc(N * M * sizeof(int));
    h_b = (int *)malloc(N * M * sizeof(int));
    h_c = (int *)malloc(N * M * sizeof(int));


	hipMalloc((void**)&d_a, sizeof(int)*M*N);
	hipMalloc((void**)&d_b, sizeof(int)*M*N);
	hipMalloc((void**)&d_c, sizeof(int)*M*N);

	for (int i = 0; i < N * M; i++) {
        // h_a[i] = 1.0f;
        // h_b[i] = 2.0f;
        h_a[i] = rand()%100;
        h_b[i] = rand()%100;
    }
    
    printFn(h_a,'A');
    
    printFn(h_b,'B');

	hipMemcpy(d_a,h_a, sizeof(int)*M*N, hipMemcpyHostToDevice );
	hipMemcpy(d_b,h_b, sizeof(int)*M*N, hipMemcpyHostToDevice );

	dim3 gridDim((N+15)/16,(M+15)/16);
	dim3 blockDim(16,16);


	addMatrixRow<<<M, 1>>>(d_a,d_b,d_c);
	hipMemcpy(h_c,d_c, sizeof(int)*M*N, hipMemcpyDeviceToHost );
	printFn(h_c,'C');
	hipFree(d_c);
	hipMalloc((void**)&d_c, sizeof(int)*M*N);


	addMatrixCol<<<1, N>>>(d_a,d_b,d_c);
	hipMemcpy(h_c,d_c, sizeof(int)*M*N, hipMemcpyDeviceToHost );
	printFn(h_c,'C');
	hipFree(d_c);
	hipMalloc((void**)&d_c, sizeof(int)*M*N);



	addMatrixEle<<<gridDim,blockDim>>>(d_a,d_b,d_c);
	hipMemcpy(h_c,d_c, sizeof(int)*M*N, hipMemcpyDeviceToHost );
	hipFree(d_c);
	printFn(h_c,'C');


	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	return 0;
	// cudaMalloc((void**)&d_c, sizeof(int)*M*N);



}
