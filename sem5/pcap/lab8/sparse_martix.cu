#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Define the number of threads per block
#define THREADS_PER_BLOCK 256

// Function to perform SpMV using CSR format
__global__ void spmv_csr_kernel(const float* values, const int* column_indices,
                                const int* row_offsets, const float* vector,
                                float* result, int num_rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows) {
        float dot_product = 0.0f;
        int row_start = row_offsets[row];
        int row_end = row_offsets[row + 1];
        
        for (int j = row_start; j < row_end; ++j) {
            int col = column_indices[j];
            dot_product += values[j] * vector[col];
        }
        
        result[row] = dot_product;
    }
}

int main() {
    int num_rows, num_cols, num_nonzeros;

    // Get dimensions and number of non-zero elements from the user
    printf("Enter the number of rows: ");
    scanf("%d", &num_rows);
    printf("Enter the number of columns: ");
    scanf("%d", &num_cols);
    printf("Enter the number of non-zero elements: ");
    scanf("%d", &num_nonzeros);

    // Allocate memory for CSR arrays on the host
    float* values = (float*)malloc(sizeof(float) * num_nonzeros);
    int* column_indices = (int*)malloc(sizeof(int) * num_nonzeros);
    int* row_offsets = (int*)malloc(sizeof(int) * (num_rows + 1));

    // Get user input for CSR arrays
    printf("Enter the non-zero values (one per line):\n");
    for (int i = 0; i < num_nonzeros; ++i) {
        scanf("%f", &values[i]);
    }
    
    printf("Enter the column indices (one per line):\n");
    for (int i = 0; i < num_nonzeros; ++i) {
        scanf("%d", &column_indices[i]);
    }
    
    printf("Enter the row offsets (one per line, including 0 at the beginning and the end):\n");
    for (int i = 0; i <= num_rows; ++i) {
        scanf("%d", &row_offsets[i]);
    }

    // Allocate memory for the vector on the host
    float* vector = (float*)malloc(sizeof(float) * num_cols);

    // Get user input for the vector
    printf("Enter the vector elements (one per line):\n");
    for (int i = 0; i < num_cols; ++i) {
        scanf("%f", &vector[i]);
    }

    // Allocate memory for the result on the host
    float* result = (float*)malloc(sizeof(float) * num_rows);

    // Allocate memory for CSR arrays and the vector on the device
    float* d_values;
    int* d_column_indices;
    int* d_row_offsets;
    float* d_vector;
    float* d_result;

    hipMalloc((void**)&d_values, sizeof(float) * num_nonzeros);
    hipMalloc((void**)&d_column_indices, sizeof(int) * num_nonzeros);
    hipMalloc((void**)&d_row_offsets, sizeof(int) * (num_rows + 1));
    hipMalloc((void**)&d_vector, sizeof(float) * num_cols);
    hipMalloc((void**)&d_result, sizeof(float) * num_rows);

    // Copy CSR arrays and vector from host to device
    hipMemcpy(d_values, values, sizeof(float) * num_nonzeros, hipMemcpyHostToDevice);
    hipMemcpy(d_column_indices, column_indices, sizeof(int) * num_nonzeros, hipMemcpyHostToDevice);
    hipMemcpy(d_row_offsets, row_offsets, sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, sizeof(float) * num_cols, hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int num_blocks = (num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv_csr_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_values, d_column_indices, d_row_offsets, d_vector, d_result, num_rows);

    // Copy the result from device to host
    hipMemcpy(result, d_result, sizeof(float) * num_rows, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result vector:\n");
    for (int i = 0; i < num_rows; ++i) {
        printf("%f\n", result[i]);
    }

    // Free device memory
    hipFree(d_values);
    hipFree(d_column_indices);
    hipFree(d_row_offsets);
    hipFree(d_vector);
    hipFree(d_result);

    // Clean up host memory
    free(values);
    free(column_indices);
    free(row_offsets);
    free(vector);
    free(result);

    return 0;
}
