#include <stdio.h>
#include <hip/hip_runtime.h>

// Function to replace each row with elements raised to a power
__global__ void modify_matrix_kernel(float* matrix, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M) {
        for (int col = 0; col < N; ++col) {
            int index = row * N + col;
            int power = row + 1; // Power for this row (1 for the first row, 2 for the second row, and so on)
            
            // Raise the element to the power
            float element = matrix[index];
            float powered_element = 1.0;
            for (int i = 0; i < power; ++i) {
                powered_element *= element;
            }
            
            matrix[index] = powered_element;
        }
    }
}

int main() {
    int M, N;

    // Get matrix dimensions (M and N) from the user
    printf("Enter the number of rows (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns (N): ");
    scanf("%d", &N);

    // Allocate memory for the matrix on the host
    float* matrix = (float*)malloc(sizeof(float) * M * N);

    // Get user input for the matrix
    printf("Enter the elements of the matrix:\n");
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            scanf("%f", &matrix[i * N + j]);
        }
    }

    // Allocate memory for the matrix on the device
    float* d_matrix;
    hipMalloc((void**)&d_matrix, sizeof(float) * M * N);

    // Copy the matrix from host to device
    hipMemcpy(d_matrix, matrix, sizeof(float) * M * N, hipMemcpyHostToDevice);

    // Launch the CUDA kernel to modify the matrix
    int num_threads = 256;
    int num_blocks = (M + num_threads - 1) / num_threads;
    modify_matrix_kernel<<<num_blocks, num_threads>>>(d_matrix, M, N);

    // Copy the modified matrix from device to host
    hipMemcpy(matrix, d_matrix, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Print the modified matrix
    printf("Modified Matrix:\n");
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", matrix[i * N + j]);
        }
        printf("\n");
    }

    // Free device and host memory
    hipFree(d_matrix);
    free(matrix);

    return 0;
}
