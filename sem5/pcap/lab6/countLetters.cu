// #include <stdio.h>
// #include <stdlib.h>
// #include <string.h>

// __global__ void countWordOccurrences(const char* sentence,char targetWord, int* result,int n) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     // Iterate through the sentence and count word occurrences
//     if (sentence[tid]==targetWord)
//     {
//         atomicAdd(result, 1);
//     }
// }

// int main() {
//     const char* sentence = "this is a test sentence. this sentence contains the word test multiple times. Test, test, test!";
//     char targetWord = 't';
//     int* d_result; // Device variable for result
//     int result ; // Host variable for result


//     // Allocate memory on the device for the result
//     cudaMalloc((void**)&d_result, sizeof(int));
//     cudaMemcpy(d_result, &result, sizeof(int), cudaMemcpyHostToDevice);
//     int n = strlen(sentence);

//     // Define thread and block configurations
//     int blockSize = 256;
//     int numBlocks = (n) / blockSize;

//     // Launch the CUDA kernel
//     countWordOccurrences<<<numBlocks, blockSize>>>(sentence, targetWord, d_result,n);

//     // Copy the result back to the host
//     cudaMemcpy(&result, d_result, sizeof(int), cudaMemcpyDeviceToHost);
    
//     // Print the result
//     printf("The word '%c' appears %d times in the sentence.\n", targetWord,result );

//     // // Copy the result back to the host
//     // cudaMemcpy(&result, d_result, sizeof(int), cudaMemcpyDeviceToHost);

//     // Free device memory
//     cudaFree(d_result);

    

//     return 0;
// }




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_SENTENCE_LENGTH 1024
#define MAX_WORD_LENGTH 32

__global__ void countWordOccurrences(const char* sentence, const char* targetWord, int* d_result, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate through the sentence and count word occurrences
    if (tid < n) {
        bool match = true; // Assume a match
        for (int i = 0; i < n; i++) {
            if (sentence[tid + i] != targetWord[i]) {
                match = false; // No match
                break;
            }
        }
        if (match) {
            atomicAdd(d_result, 1); // Increment the result atomically
            tid += n; // Skip the matched word
        } 
            else {
            tid++;
        }
    // printf("%3d \n",*d_result );
    }
}

int main() {
    const char* sentence = "This is a test sentence. This sentence contains the word test multiple times. Test, test, test!";
    const char* targetWord = "test";

    int* d_result; // Device variable for result
    int result  = 0; // Host variable for result
    int ans;
    int n = strlen(targetWord);
    // Allocate memory on the device for the result
    hipMalloc((void**)&d_result, sizeof(int));
    hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

    // Define thread and block configurations
    int blockSize = 16;
    int numBlocks = (strlen(sentence) + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    countWordOccurrences<<<numBlocks, blockSize>>>(sentence, targetWord, d_result,n);

    // Copy the result back to the host
    hipMemcpy(&ans, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_result);

    // Print the result
    printf("The word '%s' appears %d times in the sentence.\n", targetWord, ans);

    return 0;
}