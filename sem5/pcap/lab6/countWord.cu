// #include <stdio.h>
// #include <stdlib.h>
// #include <string.h>

// #define MAX_SENTENCE_LENGTH 1024
// #define MAX_WORD_LENGTH 32

// __global__ void countWordOccurrences(const char* sentence, const char* targetWord, int* d_result, int n) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     // Iterate through the sentence and count word occurrences
//     while (tid < n) {
//         bool match = true; // Assume a match
//         for (int i = 0; i < n; i++) {
//             if (sentence[tid + i] != targetWord[i]) {
//                 match = false; // No match
//                 break;
//             }
//         }
//         if (match) {
//             atomicAdd(d_result, 1); // Increment the result atomically
//             tid += n; // Skip the matched word
//         } 
//             else {
//             tid++;
//         }
//     // printf("%3d \n",*d_result );
//     }
// }

// int main() {
//     const char* sentence = "This is a test sentence. This sentence contains the word test multiple times. Test, test, test!";
//     const char* targetWord = "test";

//     int* d_result; // Device variable for result
//     int result  = 0; // Host variable for result
//     int ans;
//     int n = strlen(targetWord);
//     // Allocate memory on the device for the result
//     cudaMalloc((void**)&d_result, sizeof(int));
//     cudaMemcpy(d_result, &result, sizeof(int), cudaMemcpyHostToDevice);

//     // Define thread and block configurations
//     int blockSize = 32;
//     int numBlocks = (strlen(sentence) + blockSize - 1) / blockSize;

//     // Launch the CUDA kernel
//     countWordOccurrences<<<numBlocks, blockSize>>>(sentence, targetWord, d_result,n);

//     // Copy the result back to the host
//     cudaMemcpy(&ans, d_result, sizeof(int), cudaMemcpyDeviceToHost);

//     // Free device memory
//     cudaFree(d_result);

//     // Print the result
//     printf("The word '%s' appears %d times in the sentence.\n", targetWord, ans);

//     return 0;
// }
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

/* 
write a program in cuda to read a sentence with equal length words, 
count numner of times a word is repeated, use atomin functions
*/

__global__ void CUDACount(char* A, char* B, int* len, int* wordLen, int* cnt){
    int idx = threadIdx.x, flag=1;
    
    if(idx + *wordLen <= *len){
        for(int i=0; i<*wordLen; i++){
            if(A[idx + i] != B[i]){
                flag = 0;
                break;
            }
        }

        if(flag==1)
            atomicAdd(cnt, 1);
    }
}

int main(){
    char A[N], B[N];
    char *d_A, *d_B;

    int count=0, len, wordLen,res;
    int *d_count, *d_len, *d_wordLen;

    hipError_t err = hipGetLastError();

    printf("Enter String : ");
    scanf("%[^\n]%*c", A);
    printf("String : %s\n\n", A);

    printf("Enter Word to be searched in String : ");
    scanf("%s", B);
    printf("Word : %s\n\n", B);

    len = strlen(A);
    wordLen = strlen(B);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 1 : %s\n", hipGetErrorString(err));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 2 : %s\n", hipGetErrorString(err));

    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_B, strlen(B)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    hipMalloc((void**)&d_len, sizeof(int));
    hipMalloc((void**)&d_wordLen, sizeof(int));
    hipMalloc((void**)&res, sizeof(int));

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3 : %s\n", hipGetErrorString(err));

    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLen, &wordLen, sizeof(int), hipMemcpyHostToDevice);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3.5 : %s\n", hipGetErrorString(err));
        
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, strlen(B)*sizeof(char), hipMemcpyHostToDevice);


    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 4 : %s\n", hipGetErrorString(err));

    CUDACount<<<1, strlen(A)>>>(d_A, d_B, d_len, d_wordLen, d_count);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 5 : %s\n", hipGetErrorString(err));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    

    hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Occurances of '%s' = %d\n", B, res);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_len);
    hipFree(d_wordLen);
    hipFree(d_count);
    return 0;
}