
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
// #include<cuda_runtime.h>
#include<string.h>

__global__ void countLetter(char* sentense,  char letter,int *d_res, int n ){
	int idx = blockIdx.x* blockIdx.x + threadIdx.x;
	if(idx<n){
		if(sentense[idx]==letter){
			atomicAdd(d_res,1);
		}
	}
}

int main()
{
	char *h_sentense = "This is a test, to test the testing code for testing cuda test";
	const char h_letter  = 't';
	// char *d_letter;
	int *h_res = 0;
	int *d_res;
	char *d_sentense;

	//Allocate memory for kernel variables
	hipMalloc((void **)d_sentense, strlen(h_sentense)*sizeof(char));
	hipMalloc((void **)d_res, sizeof(int));
	// cudaMalloc(d_letter, sizeof(char));

	//Copy the values to kernel variables
	hipMemcpy(d_sentense,h_sentense, strlen(h_sentense)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_res,h_res, sizeof(int),hipMemcpyHostToDevice);
	// cudaMemcpy(d_letter,h_letter, sizeof(int),cudaMemcpyHostToDevice);

	dim3 gridDim(strlen(h_sentense),1,1);
	dim3 blockDim(256,1,1);
	countLetter<<<gridDim, blockDim>>>(d_sentense, h_letter,d_res ,strlen(h_sentense));

	//copy the result to host variables
	hipMemcpy(h_res,d_res,sizeof(int), hipMemcpyDeviceToHost);


	printf("letter %c occurs %d times in : '%s'\n",h_letter, *h_res , h_sentense );
	//Free up the device memory
	hipFree(d_res);
	// cudaFree(d_letter);
	hipFree(d_sentense);

	//free host varables memory
	free(h_sentense);
	free(h_res);

	return 0;
	
}