
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMul(int* A, int* B, int* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        int sum = 0;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

int main() {
    int m = 2; // Number of rows in matrix A
    int n = 3; // Number of columns in matrix A and rows in matrix B
    int k = 2; // Number of columns in matrix B
    int* A, * B, * C;
    int *d_A, *d_B, *d_C;

    // Dynamically allocate memory for matrices A, B, and C
    A = (int*)malloc(m * n * sizeof(int));
    B = (int*)malloc(n * k * sizeof(int));
    C = (int*)malloc(m * k * sizeof(int));

    // Initialize matrices A and B with your data
    printf("Enter row of dim %d X %d:\n",m,n );
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d", &A[i*m+j]);
        }
    }
    printf("Enter row of dim %d X %d:\n",n,k );
    for(int i=0;i<n;i++){
        for(int j=0;j<k;j++){
            scanf("%d", &B[i*n+j]);
        }
    }
    // Allocate memory on the device
    hipMalloc((void**)&d_A, m * n * sizeof(int));
    hipMalloc((void**)&d_B, n * k * sizeof(int));
    hipMalloc((void**)&d_C, m * k * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(k , m);
    dim3 dimBlock(16, 16);

    // Launch the matrix multiplication kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, k);

    // Copy the result matrix C from device to host
    hipMemcpy(C, d_C, m * k * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result matrix C
    printf("This is matrix C: \n");
    for(int i=0;i<m;i++){
        for (int j = 0; j < k; j++)
        {
            printf("%3d",C[i*m+j] );
        }
        printf("\n");
    }

    // Free dynamically allocated memory
    free(A);
    free(B);
    free(C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
