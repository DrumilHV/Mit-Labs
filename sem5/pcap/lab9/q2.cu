
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MASK_WIDTH 3 // Define the mask dimensions (e.g., 3x3 mask)

__global__ void convolution2D(int* input, int* mask, int* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;

    for (int i = 0; i < MASK_WIDTH; i++) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            int mask_row = i - (MASK_WIDTH / 2);
            int mask_col = j - (MASK_WIDTH / 2);

            int image_row = row + mask_row;
            int image_col = col + mask_col;

            // Check if the pixel is within the image bounds
            if (image_row >= 0 && image_row < height && image_col >= 0 && image_col < width) {
                sum += input[image_row * width + image_col] * mask[i * MASK_WIDTH + j];
            }
        }
    }

    output[row * width + col] = sum;
}

int main() {
    int width = 4;   // Width of the input and output arrays
    int height = 4;  // Height of the input and output arrays

    int input[width][height]; // Input array
    for(int i=0;i<width;i++){
        for(int j=0;j<height;j++){
            input[i][j] = i*width+j;
        }
    }
    printf("the matrix A: \n");
    for(int i=0;i<width;i++){
        for(int j=0;j<height;j++){
            printf("%2d ", input[i][j]);
        }
        printf("\n");
    }

    int mask[3][3] = {{-1, -1, -1},
                      {-1,  8, -1},
                      {-1, -1, -1}}; // 3x3 convolution mask
    printf("the Mask: \n");
    for(int i=0;i<3;i++){
        for(int j=0;j<3;j++){
            printf("%2d ", mask[i][j]);
        }
        printf("\n");
    }


    int output[width][height]; // Output array

    int *d_input, *d_mask, *d_output; // Device pointers

    // Allocate memory on the device
    hipMalloc((void**)&d_input, width * height * sizeof(int));
    hipMalloc((void**)&d_mask, MASK_WIDTH * MASK_WIDTH * sizeof(int));
    hipMalloc((void**)&d_output, width * height * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_input, input, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, MASK_WIDTH * MASK_WIDTH * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(width, height); // 2D grid
    dim3 dimBlock(1, 1);        // 2D block with one thread in each dimension

    // Launch the convolution kernel
    convolution2D<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output, width, height);

    // Copy the result back from device to host
    hipMemcpy(output, d_output, width * height * sizeof(int), hipMemcpyDeviceToHost);

    // Print the output array
    printf("THe output Matrix :\n");
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%3d ", output[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}
